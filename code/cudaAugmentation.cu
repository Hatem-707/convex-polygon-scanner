#include "hip/hip_runtime.h"
/* graham_scan_cuda.cu */

#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <ctime>
#include <mutex>

using namespace std;

// --- Device-side data structures & routines ---

struct Point { int x, y; };

// Device cross product/orientation
__device__ inline int crossDev(const Point &a, const Point &b, const Point &c) {
    // (b - a) x (c - b)
    return (b.y - a.y) * (c.x - b.x)
         - (b.x - a.x) * (c.y - b.y);
}

// Device Graham scan: returns hull size
__device__ int grahamScanDevice(Point *pts, int n) {
    // 1) find pivot: lowest y, if tie highest x
    int minIdx = 0;
    for (int i = 1; i < n; ++i) {
        if (pts[i].y < pts[minIdx].y ||
           (pts[i].y == pts[minIdx].y && pts[i].x > pts[minIdx].x)) {
            minIdx = i;
        }
    }
    // swap pivot to index 0
    Point tmp = pts[0]; pts[0] = pts[minIdx]; pts[minIdx] = tmp;
    Point pivot = pts[0];

    // 2) simple insertion sort by polar angle around pivot
    //    (thrust::sort isn't directly usable with device lambda here in all cases)
    for (int i = 2; i < n; ++i) {
        for (int j = i; j > 1; --j) {
            int o = (pts[j].y - pivot.y) * (pivot.x - pts[j-1].x)
                  - (pts[j-1].y - pivot.y) * (pivot.x - pts[j].x);
            if (o < 0 || (o == 0 &&
                (pts[j].x - pivot.x)*(pts[j].x - pivot.x) + (pts[j].y - pivot.y)*(pts[j].y - pivot.y)
              < (pts[j-1].x - pivot.x)*(pts[j-1].x - pivot.x) + (pts[j-1].y - pivot.y)*(pts[j-1].y - pivot.y))) {
                // swap
                Point ctmp = pts[j]; pts[j] = pts[j-1]; pts[j-1] = ctmp;
            } else break;
        }
    }

    // 3) build hull in local stack
    extern __shared__ Point hull[];  // allocate at kernel launch: sharedMem = sizeof(Point)*n
    int m = 0;
    hull[m++] = pts[0];
    hull[m++] = pts[1];
    for (int i = 2; i < n; ++i) {
        while (m > 1 && crossDev(hull[m-2], hull[m-1], pts[i]) < 0) {
            --m;
        }
        hull[m++] = pts[i];
    }
    return m;
}

// Kernel: process one candidate polygon per thread
__global__ void grahamScanMultiKernel(Point *allPts, int *offsets, int *counts, bool *results, int target) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // each thread handles combination idx
    Point *pts = allPts + offsets[idx];
    int n = counts[idx];
    // allocate shared mem for 'hull' of size n
    int hullSize = grahamScanDevice(pts, n);
    results[idx] = (hullSize != target);
}

// --- Host-side wrapper for multi-point examination using CUDA ---

bool examineNewPointMultiCUDA(pair<int,int> newPoint, const vector<pair<int,int>> &prevSet, int targetPoly) {
    // early colinearity/duplication checks (host)
    for (auto &p1 : prevSet) {
        for (auto &p2 : prevSet) {
            if ((crossProduct({newPoint.first,newPoint.second}, p1, p2) == 0 && p1 != p2)
             || newPoint == p1 || newPoint == p2)
                return false;
        }
    }

    int setSize = (int)prevSet.size();
    vector<bool> comb(setSize, false);
    fill(comb.begin(), comb.begin() + targetPoly - 1, true);

    const int batchSize = 128;  // tune for your GPU
    vector<Point> flatPts; flatPts.reserve(batchSize * targetPoly);
    vector<int>    offsets;    offsets.reserve(batchSize);
    vector<int>    counts;     counts.reserve(batchSize);
    vector<bool>   results;

    // device buffers (will be re-used)
    Point *d_allPts = nullptr;
    int   *d_offsets = nullptr, *d_counts = nullptr;
    bool  *d_results = nullptr;
    int    maxB       = batchSize;

    size_t ptsBufBytes = sizeof(Point) * batchSize * targetPoly;
    size_t offBytes    = sizeof(int) * batchSize;
    size_t resBytes    = sizeof(bool) * batchSize;

    hipMalloc(&d_allPts, ptsBufBytes);
    hipMalloc(&d_offsets, offBytes);
    hipMalloc(&d_counts, offBytes);
    hipMalloc(&d_results, resBytes);

    bool globalOk = true;
    do {
        // build one combination
        vector<pair<int,int>> poly;
        poly.reserve(targetPoly);
        for (int i = 0; i < setSize; ++i)
            if (comb[i]) poly.push_back(prevSet[i]);
        poly.push_back(newPoint);

        // flatten
        offsets.push_back((int)flatPts.size());
        counts .push_back((int)poly.size());
        for (auto &p : poly)
            flatPts.push_back({p.first, p.second});

        if ((int)offsets.size() == batchSize) {
            // copy to device
            hipMemcpy(d_allPts,   flatPts.data(),      flatPts.size()*sizeof(Point), hipMemcpyHostToDevice);
            hipMemcpy(d_offsets,  offsets.data(),      offsets.size()*sizeof(int),   hipMemcpyHostToDevice);
            hipMemcpy(d_counts,   counts.data(),       counts.size()*sizeof(int),    hipMemcpyHostToDevice);

            // launch
            int threads = 128;
            int blocks  = (batchSize + threads - 1) / threads;
            size_t sharedMemBytes = sizeof(Point) * targetPoly;
            grahamScanMultiKernel<<<blocks,threads,sharedMemBytes>>>(
                d_allPts, d_offsets, d_counts, d_results, targetPoly
            );
            hipDeviceSynchronize();

            // retrieve
            results.resize(batchSize);
            hipMemcpy(results.data(), d_results, resBytes, hipMemcpyDeviceToHost);

            // check
            for (bool ok : results) if (!ok) { globalOk = false; break; }
            if (!globalOk) break;

            // reset
            flatPts.clear(); offsets.clear(); counts.clear();
        }

        if (!globalOk) break;
    } while (prev_permutation(comb.begin(), comb.end()));

    // process any leftovers
    if (globalOk && !offsets.empty()) {
        int smallN = (int)offsets.size();
        hipMemcpy(d_allPts, flatPts.data(), flatPts.size()*sizeof(Point), hipMemcpyHostToDevice);
        hipMemcpy(d_offsets, offsets.data(), smallN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_counts, counts.data(),   smallN*sizeof(int), hipMemcpyHostToDevice);
        int threads = 64;
        int blocks  = (smallN + threads - 1) / threads;
        size_t sharedMemBytes = sizeof(Point) * targetPoly;
        grahamScanMultiKernel<<<blocks,threads,sharedMemBytes>>>(d_allPts, d_offsets, d_counts, d_results, targetPoly);
        hipDeviceSynchronize();
        vector<bool> smallRes(smallN);
        hipMemcpy(smallRes.data(), d_results, smallN*sizeof(bool), hipMemcpyDeviceToHost);
        for (bool ok : smallRes) if (!ok) { globalOk = false; break; }
    }

    hipFree(d_allPts);
    hipFree(d_offsets);
    hipFree(d_counts);
    hipFree(d_results);

    return globalOk;
}

// --- Main (adjusted to call CUDA variant) ---
int main(int argc, const char* argv[]) {
    const int n = 13;
    const int h = 6;
    const int x = 200;
    const int y = 200;
    srand((unsigned)time(0));
    const string saveFile = "1points.txt";

    int toBeTested = 100;
    int successes  = 0;

    time_t start = time(0);

    while (toBeTested--) {
        long long iterations = 0;
        vector<pair<int,int>> emptySet;
        bool found = false;
        mutex mtx;
        vector<thread> pool;

        // generate initial emptySet via your threadFunctionEmptySet CPU code
        for (int i = 0; i < 14; ++i) {
            pool.emplace_back(
                threadFunctionEmptySet,
                &mtx, n, h, x, y, &emptySet, &found, &iterations
            );
        }
        for (auto &t : pool) t.join();

        cout << "Seed set: ";
        for (auto &p : emptySet) cout << p.first << "," << p.second << " ";
        cout << '\n';

        long long incr = 0;
        int valid = 0;
        const int toAdd = 3;
        for (int added = 0; added < toAdd; ++added) {
            if (added > valid) break;
            for (int i = 0; i < x; ++i) {
                for (int j = 0; j < y; ++j) {
                    ++incr;
                    if (examineNewPointMultiCUDA({i,j}, emptySet, h)) {
                        ++valid;
                        emptySet.emplace_back(i,j);
                    }
                    if (valid == toAdd) break;
                }
                if (valid == toAdd) break;
            }
        }

        // cout << format("Extend iterations: {} --> size {}\n", incr, emptySet.size());

        if (valid == toAdd) {
            auto confHull = checkPointsForPolygonH((int)emptySet.size(), h, emptySet);
            if (confHull.size() == 1) {
                ++successes;
                cout << "Confirmed!\n";
                ofstream f(saveFile, ios::app);
                for (auto &p : emptySet)
                    f << p.first << " " << p.second << "|";
                f << '\n';
            } else {
                cout << "Mistake!\n";
            }
        } else {
            cout << "Failed to extend to size " << toAdd << "\n";
        }

        cout << "Runs left: " << toBeTested << ", Successes: " << successes << "\n";
    }

    time_t end = time(0);

    cout<<"Time: "<<end-start<<endl

    return 0;
}
